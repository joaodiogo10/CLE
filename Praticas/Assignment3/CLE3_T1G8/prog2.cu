#include "hip/hip_runtime.h"
#include "common.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <unistd.h>

// (row, col, order)
#define idx(x,y,order)(x*order+y)

#define KRED  "\x1B[31m"
#define KGRN  "\x1B[32m"
#define KNRM  "\x1B[0m"

inline void switch_row(double *matrix, int row1, int row2, int order);

void determinantOnHostColumns(double *matrices, int numberOfMatrix, double *determinant, int order);

__global__ void determinantOnGPUColumns(double *mat, double *determinant, int order);

void checkResult(double *cpuRef, double *gpuRef, int nDeterminants);

static double get_delta_time(void);

int main(int argc, char **argv)
{
    printf("%s Starting...\n", argv[0]);

    // process cli 
    int opt;
    char * fileName;

    do {
        switch((opt = getopt(argc, argv, "f:h"))) {
            case 'f':
                fileName = optarg;
                break;
                
            case 'h':
                printf("-f      --- filename\n");
                break;
        }
    }
    while(opt != -1);

    // set up device
    int dev = 0;
    hipDeviceProp_t deviceProp;
    CHECK(hipGetDeviceProperties(&deviceProp, dev));
    printf("Using Device %d: %s\n", dev, deviceProp.name);
    CHECK(hipSetDevice(dev));

    // set up data size of matrix
    int order;
    int numberOfMatrix;
    
    FILE * ptrFile = fopen(fileName, "r"); 
    if(ptrFile == NULL) {
        fprintf(stderr,"Error opening file");
        printf("%s\n", fileName);
        exit(EXIT_FAILURE);
    }     
    size_t size = fread(&numberOfMatrix, sizeof(unsigned int), 1, ptrFile);
    if(size != 1)
    {
        fprintf(stderr,"Error reading the number of matrix in the file\n");
        exit(EXIT_FAILURE);
    }

    size = fread(&order, sizeof(unsigned int), 1, ptrFile);
    if(size != 1)
    {
        fprintf(stderr,"Error reading order from file\n");
        exit(EXIT_FAILURE);
    }

    printf("Filename: %s\nNumber of matrices: %d\nMatrices order: %d\n", fileName, numberOfMatrix, order);

    int nBytesMatrices = order * order * numberOfMatrix * sizeof(double);
    int nBytesDeterminants = numberOfMatrix * sizeof(double);
    
    if ((nBytesMatrices + nBytesDeterminants) > (size_t) 5e9)
    { 
        fprintf (stderr,"The GeForce GTX 1660 Ti cannot handle more than 5GB of memory!\n");
        exit(EXIT_FAILURE);
    }
    
    printf ("Total matrices data size: %d\n", nBytesMatrices);
    printf ("Total determinants data size: %d\n", nBytesDeterminants);

    //host memory
    double determinantRefCPU[numberOfMatrix];
    double determinantRefGPU[numberOfMatrix];
    double *h_matrices = (double *)malloc(nBytesMatrices);
    size = fread(h_matrices, sizeof(double), order * order * numberOfMatrix, ptrFile);
    if(size != order * order * numberOfMatrix)
    {
        fprintf(stderr,"Error matrices from file\n");
        exit(EXIT_FAILURE);
    }


    // malloc device global memory
    double *d_matrices;
    double *d_determinant;

    CHECK(hipMalloc((void **)&d_matrices, nBytesMatrices));
    CHECK(hipMalloc((void **)&d_determinant, nBytesDeterminants));

    // transfer data from host to device
    (void) get_delta_time();
    CHECK(hipMemcpy(d_matrices, h_matrices, nBytesMatrices, hipMemcpyHostToDevice));
    printf ("The transfer of %d bytes from the host to the device took %.3e seconds\n",
            nBytesMatrices + nBytesDeterminants, get_delta_time());

    // calculate determinant at host side
    (void) get_delta_time();
    determinantOnHostColumns(h_matrices, numberOfMatrix, determinantRefCPU, order);
    printf("The cpu kernel took %.3e seconds to run (single core)\n", get_delta_time ());

    unsigned int gridDimX, gridDimY, blockDimX, blockDimY;

    // invoke kernel at host side
    blockDimX = order;
    blockDimY = 1 << 0;
    gridDimX = numberOfMatrix;
    gridDimY = 1 << 0;
    dim3 block(blockDimX, blockDimY);
    dim3 grid(gridDimX, gridDimY);

    (void) get_delta_time ();
    determinantOnGPUColumns<<<grid, block, order * sizeof(double)>>>(d_matrices, d_determinant, order);
    CHECK(hipDeviceSynchronize());
    CHECK(hipGetLastError());
    printf("determinantOnGPUCols <<<(%d,%d), (%d,%d)>>> elapsed %.3e sec\n", grid.x,
           grid.y,
           block.x, block.y, get_delta_time ());


    // copy kernel result back to host side
    CHECK(hipMemcpy(determinantRefGPU, d_determinant, nBytesDeterminants, hipMemcpyDeviceToHost));

    // check device results
    checkResult(determinantRefCPU, determinantRefGPU, numberOfMatrix);

    // free device global memory
    CHECK(hipFree(d_matrices));
    CHECK(hipFree(d_determinant));

    // reset device
    CHECK(hipDeviceReset());

    return (0);
}

inline void switch_row(double *matrix, int row1, int row2, int order) {
    double aux;
    
    for(int i = 0; i < order; i++) 
    {
        aux = matrix[idx(row1,i,order)];
        matrix[idx(row1,i,order)] = matrix[idx(row2,i,order)];
        matrix[idx(row2,i,order)] = aux;
    }
}

void determinantOnHostColumns(double *matrices, int numberOfMatrix, double *determinant, int order)
{
    int sign = 1;
    double ratio = 1;
    
    for(int n = 0; n < numberOfMatrix; n++)
    {
        double *matrix = &matrices[n * order * order];
        determinant[n] = 1;
        
        // for each row
        for(int i = 0; i < order; i++) 
        {
            // check if the row can be used, otherwise, switch that row
            if(matrix[idx(i,i,order)] == 0) {
                bool determinantIsZero = true;
                for(int j = i+1; j < order; j++) 
                {
                    if(matrix[idx(j,i,order)] != 0) 
                    {
                        switch_row(matrix, i, j, order);
                        sign = (sign == 1) ? -1: 1;
                        determinantIsZero = false;
                        break;
                    }
                }
                if(determinantIsZero)
                {
                    determinant[n] = 0;
                    break;
                }                       
            }

            // for all other rows
            for(int j = i + 1; j < order; j++) {
                ratio = matrix[idx(j,i,order)] / matrix[idx(i,i,order)];
                for(int k = 0; k < order; k++) {
                    matrix[idx(j,k,order)] = matrix[idx(j,k,order)] - ratio * matrix[idx(i,k,order)];
                }
            }
            determinant[n] *= matrix[idx(i,i,order)];
        }
        determinant[n] *= sign;
    }
}

__global__ void determinantOnGPUColumns(double *mat, double *determinant, int order)
{
    extern __shared__ double tmp[];  

    unsigned int columnNumber = threadIdx.x + threadIdx.y * blockDim.x;
    unsigned int matrixNumber = blockIdx.x + blockIdx.y * gridDim.x;
    unsigned int size = order * order;

    double *matrix = &mat[matrixNumber * size];

    int sign = 1;
    determinant[matrixNumber] = 1;
    
    //for each row
    for(int i = 0; i < order; i++)
    {
        //swap row if necessary
        if(matrix[idx(i,i,order)] == 0) {
            bool determinantIsZero = true;
            for(int j= i + 1; j < order; j++) {
                if(matrix[idx(j,i,order)] != 0) {
                    double t = matrix[idx(i,columnNumber,order)];
                    matrix[idx(i,columnNumber,order)] = matrix[idx(j,columnNumber,order)];
                    matrix[idx(j,columnNumber,order)] = t;
                    sign = (sign == 1) ? -1 : 1;
                    determinantIsZero = false;

                    break;
                }
            } 
            //determinant is 0
            if(determinantIsZero)
            {            
                if(columnNumber == 0)
                    determinant[matrixNumber] = 0;
                return;               
            }
        }
        __syncthreads();

        //Read all necessary values
        tmp[columnNumber] = matrix[idx(columnNumber,i,order)];
        __syncthreads();

        //For all other rows
        for(int j = i + 1; j < order; j++)
        {
            double ratio = tmp[j] / matrix[idx(i,i,order)];

            //process corresponding col
            matrix[idx(j,columnNumber,order)] = matrix[idx(j,columnNumber,order)]-ratio*matrix[idx(i,columnNumber,order)];
        }
        __syncthreads();
    }

    //calculate determinant
    if(columnNumber == 0)
    {
        determinant[matrixNumber] = sign;
        for(int i = 0; i < order; i++)
            determinant[matrixNumber] = determinant[matrixNumber] * matrix[idx(i,i,order)];
    }
}

void checkResult(double *cpuRef, double *gpuRef, int nDeterminants)
{
    bool match = 1;
    for(int i = 0; i < nDeterminants; i++)
    {
        double epsilon = (1 - cpuRef[i] / gpuRef[i]) * 100;
        if(epsilon < 0)
            epsilon = -epsilon;

        if (epsilon > 0.00001)
        {
            match = 0;
            printf("%sError: Matrix %3d - host %.8e \t gpu %.8e\n%s", KRED, i + 1, cpuRef[i], gpuRef[i], KNRM);
            break;
        }

        printf("%sCorrect: Matrix %3d - host %.3e \t gpu %.3e\n%s", KGRN, i + 1, cpuRef[i], gpuRef[i], KNRM);
    }

    if (match)
        printf("Determinants match.\n\n");
    else
        printf("Determinants do not match.\n\n");
}

static double get_delta_time(void)
{
  static struct timespec t0,t1;

  t0 = t1;
  if(clock_gettime(CLOCK_MONOTONIC,&t1) != 0)
  {
    perror("clock_gettime");
    exit(1);
  }
  return (double)(t1.tv_sec - t0.tv_sec) + 1.0e-9 * (double)(t1.tv_nsec - t0.tv_nsec);
}